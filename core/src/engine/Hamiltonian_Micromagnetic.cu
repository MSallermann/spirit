
#include <hip/hip_runtime.h>
#ifdef SPIRIT_USE_CUDA

#include <engine/Hamiltonian_Micromagnetic.hpp>
#include <engine/Vectormath.hpp>
#include <engine/Neighbours.hpp>
#include <data/Spin_System.hpp>
#include <utility/Constants.hpp>
#include <algorithm>
#include <complex>
#include <Eigen/Dense>
#include <Eigen/Core>
#include "FFT.hpp"
#include <cstdio>
#include <utility/Custom_Field.hpp>
using namespace Data;
using namespace Utility;
namespace C = Utility::Constants_Micromagnetic;
using Engine::Vectormath::check_atom_type;
using Engine::Vectormath::idx_from_pair;
using Engine::Vectormath::cu_check_atom_type;
using Engine::Vectormath::cu_idx_from_pair;
using Engine::Vectormath::cu_tupel_from_idx;


namespace Engine
{
	Hamiltonian_Micromagnetic::Hamiltonian_Micromagnetic(
		scalarfield external_field_magnitude, vectorfield external_field_normal,
		intfield n_anisotropies, std::vector<std::vector<scalar>>  anisotropy_magnitudes, std::vector<std::vector<Vector3>> anisotropy_normals,
		scalarfield  exchange_stiffness,
		scalarfield  dmi,
		std::shared_ptr<Data::Geometry> geometry,
		int spatial_gradient_order,
		intfield boundary_conditions,
		Vector3 cell_sizes,
		scalarfield Ms, int region_num
	) : Hamiltonian(boundary_conditions), spatial_gradient_order(spatial_gradient_order), geometry(geometry),
		external_field_magnitude(external_field_magnitude), external_field_normal(external_field_normal),
		n_anisotropies(n_anisotropies), anisotropy_magnitudes(anisotropy_magnitudes), anisotropy_normals(anisotropy_normals),
		exchange_stiffness(exchange_stiffness), dmi(dmi),
		cell_sizes(cell_sizes), Ms(Ms), region_num(region_num)
	{
		// Generate interaction pairs, constants etc.
		regions=intfield(geometry->nos,0);
		/*for (int i=0; i < ((int)(geometry->nos/2)); i++){
			regions[i]=1;
		}*/

		Regionvalues test;
		//regions_book=regionbook(region_num);
		//cudaMallocHost((void**)&regions_book, region_num*sizeof(regions_book));
		regions_book=regionbook(region_num);
		for (int i=0;i<region_num;i++){
			test.external_field_magnitude=external_field_magnitude[i];
			test.external_field_normal=external_field_normal[i];
			test.Ms=Ms[i];
			test.Dmi=dmi[i];
			test.Aexch=exchange_stiffness[i];
			test.n_anisotropies=n_anisotropies[i];
			for (int j=0;j<n_anisotropies[i];j++){
				test.anisotropy_magnitudes[j]=anisotropy_magnitudes[i][j];
				test.anisotropy_normals[j]=anisotropy_normals[i][j];
			}
			this->regions_book[i]=test;
		}
		exchange_table=std::vector<std::vector<scalar>>(region_num, std::vector<scalar>(region_num,0));
		for (int i=0;i<region_num;i++){
			for (int j=0;j<region_num;j++){
				exchange_table[i][j]=(this->regions_book[i].Aexch+this->regions_book[j].Aexch)/2;
			}
		}
		this->Update_Interactions();
	}
/*
	Hamiltonian_Micromagnetic::Hamiltonian_Micromagnetic(
		scalar external_field_magnitude, Vector3 external_field_normal,
		int n_anisotropies, scalarfield anisotropy_magnitudes, vectorfield anisotropy_normals,
		scalar exchange_constant,
		scalar dmi_constant,
		std::shared_ptr<Data::Geometry> geometry,
		int spatial_gradient_order,
		intfield boundary_conditions,
		Vector3 cell_sizes,
		scalar Ms
	) : Hamiltonian(boundary_conditions), spatial_gradient_order(spatial_gradient_order), geometry(geometry),
		external_field_magnitude(external_field_magnitude), external_field_normal(external_field_normal),
		n_anisotropies(n_anisotropies), anisotropy_magnitudes(anisotropy_magnitudes), anisotropy_normals(anisotropy_normals),
		exchange_constant(exchange_constant), dmi_constant(dmi_constant),
		cell_sizes(cell_sizes), Ms(Ms)
	{
		// Generate interaction pairs, constants etc.
		this->Update_Interactions();
	}
*/
    void Hamiltonian_Micromagnetic::Update_Interactions()
    {
        #if defined(SPIRIT_USE_OPENMP)
        // When parallelising (cuda or openmp), we need all neighbours per spin
        const bool use_redundant_neighbours = true;
        #else
        // When running on a single thread, we can ignore redundant neighbours
        const bool use_redundant_neighbours = false;
        #endif

        // TODO: make sure that the geometry can be treated with this model:
        //       - rectilinear, only one "atom" per cell
        // if( geometry->n_cell_atoms != 1 )
        //     Log(...)

        // TODO: generate neighbour information for pairwise interactions

        // TODO: prepare dipolar interactions
		neigh = pairfield(0);
		Neighbour neigh_tmp;
		neigh_tmp.i = 0;
		neigh_tmp.j = 0;
		neigh_tmp.idx_shell = 0;
		//order x -x y -y z -z xy (-x)(-y) x(-y) (-x)y xz (-x)(-z) x(-z) (-x)z yz (-y)(-z) y(-z) (-y)z results in 9 parts of Hessian 
		neigh_tmp.translations[0] = 1;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = -1;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = 1;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = -1;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = 1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = -1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 1;
		neigh_tmp.translations[1] = 1;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = -1;
		neigh_tmp.translations[1] = -1;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 1;
		neigh_tmp.translations[1] = -1;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = -1;
		neigh_tmp.translations[1] = +1;
		neigh_tmp.translations[2] = 0;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 1;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = 1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = -1;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = -1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 1;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = -1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = -1;
		neigh_tmp.translations[1] = 0;
		neigh_tmp.translations[2] = 1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = 1;
		neigh_tmp.translations[2] = 1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = -1;
		neigh_tmp.translations[2] = -1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = 1;
		neigh_tmp.translations[2] = -1;
		neigh.push_back(neigh_tmp);

		neigh_tmp.translations[0] = 0;
		neigh_tmp.translations[1] = -1;
		neigh_tmp.translations[2] = 1;
		neigh.push_back(neigh_tmp);

		this->spatial_gradient = field<Matrix3>(geometry->nos, Matrix3::Zero());
		// Update, which terms still contribute
        this->Update_Energy_Contributions();
        // Exchange tensor
		this->Prepare_Exchange();
		// Dipole-dipole (FFT)
		this->Prepare_DDI();
		external_field=vectorfield(geometry->nos,Vector3{0,0,0});

    }

    void Hamiltonian_Micromagnetic::Update_Energy_Contributions()
    {
		#ifndef SPIRIT_LOW_MEMORY
			this->energy_contributions_per_spin = std::vector<std::pair<std::string, scalarfield>>(0);
			this->gradient_contributions_per_spin = std::vector<std::pair<std::string, vectorfield>>(0);
			//this->idx_zeeman = 0;

			// External field
			//if( this->external_field_magnitude > 0 )
			//{
			this->energy_contributions_per_spin.push_back({"Zeeman", scalarfield(0)});
			this->gradient_contributions_per_spin.push_back({"Zeeman", vectorfield(0)});
			this->idx_zeeman = this->energy_contributions_per_spin.size()-1;
			//}
			//else
				//this->idx_zeeman = -1;
			// TODO: Anisotropy
			// if( ... )
			// {
			this->energy_contributions_per_spin.push_back({"Anisotropy", scalarfield(0) });
			this->gradient_contributions_per_spin.push_back({"Anisotropy", vectorfield(0) });
			this->idx_anisotropy = this->energy_contributions_per_spin.size()-1;
			// }
			// else
				//this->idx_anisotropy = -1;
			// TODO: Exchange
			// if( ... )
			// {
			this->energy_contributions_per_spin.push_back({"Exchange", scalarfield(0) });
			this->gradient_contributions_per_spin.push_back({"Exchange", vectorfield(0) });
			this->idx_exchange = this->energy_contributions_per_spin.size()-1;
			// }
			// else
				//this->idx_exchange = -1;
			// TODO: DMI
			// if( ... )
			// {
			this->energy_contributions_per_spin.push_back({"DMI", scalarfield(0) });
			this->gradient_contributions_per_spin.push_back({"DMI", vectorfield(0) });
			this->idx_dmi = this->energy_contributions_per_spin.size()-1;
			// }
			// else
				//this->idx_dmi = -1;
			// TODO: DDI
			// if( ... )
			// {
			this->energy_contributions_per_spin.push_back({"DDI", scalarfield(0) });
			this->gradient_contributions_per_spin.push_back({"DDI", vectorfield(0) });
			this->idx_ddi = this->energy_contributions_per_spin.size()-1;

			int nos = geometry->nos;
			for( auto& contrib : this->gradient_contributions_per_spin )
			{
				// Allocate if not already allocated
				if (contrib.second.size() != nos) contrib.second = vectorfield(nos, Vector3{0, 0, 0});
				// Otherwise set to zero
				else Vectormath::fill(contrib.second, Vector3{0, 0, 0});
			}
		#endif

		#ifdef SPIRIT_LOW_MEMORY
			// Energy contributions
			temp_energies=scalarfield(this->geometry->nos,0);
			energy_array=std::vector<std::pair<std::string, scalar>> (0);
			this->energy_array.push_back({"Zeeman", 0});
			this->idx_zeeman = this->energy_array.size()-1;
			this->energy_array.push_back({"Anisotropy", 0});
			this->idx_anisotropy = this->energy_array.size()-1;
			this->energy_array.push_back({"Exchange", 0});
			this->idx_exchange = this->energy_array.size()-1;
			this->energy_array.push_back({"DMI", 0 });
			this->idx_dmi = this->energy_array.size()-1;
			this->energy_array.push_back({"DDI", 0 });
			this->idx_ddi = this->energy_array.size()-1;
		#endif

    }

    void Hamiltonian_Micromagnetic::Energy_Contributions_per_Spin(const vectorfield & spins, std::vector<std::pair<std::string, scalarfield>> & contributions)
    {
		#ifndef SPIRIT_LOW_MEMORY
			if( contributions.size() != this->energy_contributions_per_spin.size() )
			{
				contributions = this->energy_contributions_per_spin;
			}

			int nos = spins.size();
			for( auto& contrib : contributions )
			{
				// Allocate if not already allocated
				if (contrib.second.size() != nos) contrib.second = scalarfield(nos, 0);
				// Otherwise set to zero
				//else Vectormath::fill(contrib.second, 0);
			}
			// External field
			if( this->idx_zeeman >=0 )     Energy_Set(spins, contributions[this->idx_zeeman].second, this->gradient_contributions_per_spin[this->idx_zeeman].second);

			// Anisotropy
			if( this->idx_anisotropy >=0 ) Energy_Set(spins, contributions[this->idx_anisotropy].second, this->gradient_contributions_per_spin[this->idx_anisotropy].second);

			// Exchange
			if( this->idx_exchange >=0 )   Energy_Set(spins, contributions[this->idx_exchange].second, this->gradient_contributions_per_spin[this->idx_exchange].second);
			// DMI
			if( this->idx_dmi >=0 )        Energy_Set(spins, contributions[this->idx_dmi].second, this->gradient_contributions_per_spin[this->idx_dmi].second);
			// DDI
			if( this->idx_ddi >=0 )        Energy_Set(spins, contributions[this->idx_ddi].second, this->gradient_contributions_per_spin[this->idx_ddi].second);
		#endif
		#ifdef SPIRIT_LOW_MEMORY
			//Energy already set in Gradient
		#endif
    }
    __global__ void CU_Energy(const Vector3 * spins, Vector3 * gradient, size_t n_cells_total, scalar * Energy, const Regionvalues * regionval, const int * regions)
	{
		//scalar m0 = (4 * 3.141592653589793238462643383279502884197169399375105820974)*1e-7;
		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
			icell < n_cells_total;
			icell += blockDim.x * gridDim.x)
		{
			Energy[icell] = 0.5 *regionval[regions[icell]].Ms* gradient[icell].dot(spins[icell]);
		}
	}
	void Hamiltonian_Micromagnetic::Energy_Set(const vectorfield & spins, scalarfield & Energy, vectorfield & gradient)
	{
		int size = geometry->n_cells_total;
		CU_Energy << <(size + 1023) / 1024, 1024 >> > (spins.data(), gradient.data(), geometry->n_cells_total,  Energy.data(), regions_book.data(), regions.data());
		CU_CHECK_AND_SYNC();
	}
	#ifdef SPIRIT_LOW_MEMORY
		__global__ void CU_Energy_Low_Memory(const Vector3 * spins, Vector3 * gradient, size_t n_cells_total, scalar * Energy, const Regionvalues * regionval, const int * regions)
		{
			//scalar m0 = (4 * 3.141592653589793238462643383279502884197169399375105820974)*1e-7;
			for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
				icell < n_cells_total;
				icell += blockDim.x * gridDim.x)
			{
				Energy[icell] = 0.5 *regionval[regions[icell]].Ms* gradient[icell].dot(spins[icell]);
			}
		}
		scalar Hamiltonian_Micromagnetic::Energy_Low_Memory(const vectorfield & spins, vectorfield & gradient)
		{
			int size = geometry->n_cells_total;
			CU_Energy_Low_Memory << <(size + 1023) / 1024, 1024 >> > (spins.data(), gradient.data(), size,  temp_energies.data(), regions_book.data(), regions.data());
			CU_CHECK_AND_SYNC();
			return Vectormath::sum(temp_energies);
		}
	#endif
    scalar Hamiltonian_Micromagnetic::Energy_Single_Spin(int ispin, const vectorfield & spins)
    {
        scalar Energy = 0;
        return Energy;
    }

    __global__ void CU_Gradient_Dummy()
    	{

    	}
    void Hamiltonian_Micromagnetic::Gradient(const vectorfield & spins, vectorfield & gradient)
    {
    	 // Set to zero
		Vectormath::fill(gradient, {0,0,0});
		this->Spatial_Gradient(spins);
		#ifndef SPIRIT_LOW_MEMORY
			Gradient_Zeeman(this->gradient_contributions_per_spin[this->idx_zeeman].second);
			Gradient_Anisotropy(spins, this->gradient_contributions_per_spin[this->idx_anisotropy].second);
			Gradient_Exchange(spins, this->gradient_contributions_per_spin[this->idx_exchange].second);
			Gradient_DMI(spins, this->gradient_contributions_per_spin[this->idx_dmi].second);
			//Gradient_DDI(spins, this->gradient_contributions_per_spin[this->idx_ddi].second);
			Vectormath::add_c_a(1,this->gradient_contributions_per_spin[this->idx_zeeman].second,gradient);
			Vectormath::add_c_a(1,this->gradient_contributions_per_spin[this->idx_anisotropy].second,gradient);
			Vectormath::add_c_a(1,this->gradient_contributions_per_spin[this->idx_exchange].second,gradient);
			Vectormath::add_c_a(1,this->gradient_contributions_per_spin[this->idx_dmi].second,gradient);
			//Vectormath::add_c_a(1,this->gradient_contributions_per_spin[this->idx_ddi].second,gradient);
		#endif
		#ifdef SPIRIT_LOW_MEMORY
			scalar temp=0;
			scalar temp1=0;
			Gradient_Zeeman(gradient);
			temp=Energy_Low_Memory(spins, gradient);
			this->energy_array[this->idx_zeeman].second=temp-temp1;
			temp1=temp;
			Gradient_Anisotropy(spins, gradient);
			temp=Energy_Low_Memory(spins, gradient);
			this->energy_array[this->idx_anisotropy].second=temp-temp1;
			//printf("%f\n", temp-temp1);
			temp1=temp;
			Gradient_Exchange(spins, gradient);
			temp=Energy_Low_Memory(spins, gradient);
			this->energy_array[this->idx_exchange].second=temp-temp1;
			//printf("%f\n", temp-temp1);
			temp1=temp;
			Gradient_DMI(spins, gradient);
			temp=Energy_Low_Memory(spins, gradient);
			this->energy_array[this->idx_dmi].second=temp-temp1;
			//printf("%f\n", temp-temp1);
			temp1=temp;
			Gradient_DDI(spins, gradient);
			temp=Energy_Low_Memory(spins, gradient);
			this->energy_array[this->idx_ddi].second=temp-temp1;
			//printf("%f\n", temp-temp1);
		#endif

		int size = geometry->n_cells_total;
		CU_Gradient_Dummy << <(size + 1023) / 1024, 1024 >> > ();
		CU_CHECK_AND_SYNC();
    }


	__global__ void CU_Gradient_Zeeman1(Vector3 * gradient, Vector3 * external_field, size_t n_cells_total, const Regionvalues * regionval, const int * regions)
	{
		scalar m0 = (4 * 3.14159265358979)*1e-7;
		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
			icell < n_cells_total;
			icell += blockDim.x * gridDim.x)
		{
				#ifndef SPIRIT_LOW_MEMORY
					gradient[icell] -= m0*regionval[regions[icell]].Ms * regionval[regions[icell]].external_field_magnitude*regionval[regions[icell]].external_field_normal;
					gradient[icell] -= m0*regionval[regions[icell]].Ms * external_field[icell];
				#endif
				#ifdef SPIRIT_LOW_MEMORY
					gradient[icell] = m0*regionval[regions[icell]].Ms * regionval[regions[icell]].external_field_magnitude*regionval[regions[icell]].external_field_normal;
					gradient[icell] -= m0*regionval[regions[icell]].Ms * external_field[icell];
				#endif
		}
	}
	/*__device__ void CustomField(Vector3 x, scalar t, Vector3 out)
		{
			if (x[0]*x[0]+x[1]*x[1]<0) {
				out = {0,0,100};
			}
			else{
				out = {0,0,0};
			}
		}
		__global__ void CU_Gradient_Zeeman_Set_External_Field(size_t n_cells_total, const Vector3 * positions, const Vector3 center, const scalar picoseconds_passed, Vector3 * external_field){
			for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
						icell < n_cells_total;
						icell += blockDim.x * gridDim.x)
			{
				CustomField(positions[icell]-center,picoseconds_passed, external_field[icell]);
			}
		}
	__device__ Vector3 CustomField(Vector3 x, scalar t)
	{
		Vector3 out={0,0,0};
		if (x[0]*x[0]+x[1]*x[1]<10000) {
			out = {0,0,0.0};
		}
		else{
			out = {0,0,-0.0};
		}
		return out;
	}

	__global__ void CU_Gradient_Zeeman_Set_External_Field(size_t n_cells_total, const Vector3 * positions, const Vector3 center, const scalar picoseconds_passed, Vector3 * external_field){
		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
					icell < n_cells_total;
					icell += blockDim.x * gridDim.x)
		{
			//Utility::Custom_Field::CustomField(positions[icell]-center,picoseconds_passed, external_field[icell]);
			external_field[icell]=CustomField(positions[icell]-center, picoseconds_passed);
		}
	}*/

	void Hamiltonian_Micromagnetic::Gradient_Zeeman(vectorfield & gradient)
	{
		int size = geometry->n_cells_total;
		Utility::Custom_Field::CustomField(size,
				this->geometry->positions.data(),
				this->geometry->center,
				picoseconds_passed,
				external_field.data());
		CU_Gradient_Zeeman1 << <(size + 1023) / 1024, 1024 >> > (gradient.data(), external_field.data(), size, regions_book.data(), regions.data());
		CU_CHECK_AND_SYNC();
	}

	__global__ void CU_Gradient_Anisotropy1(const Vector3 * spins, const int * atom_types, const int n_cell_atoms, Vector3 * gradient, size_t n_cells_total, const Regionvalues * regionval, const int * regions)
	{

		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
			icell < n_cells_total;
			icell += blockDim.x * gridDim.x)
		{
			#ifndef SPIRIT_LOW_MEMORY
				gradient[icell][0]=0;
				gradient[icell][1]=0;
				gradient[icell][2]=0;
			#endif
			for (int i = 0; i < regionval[regions[icell]].n_anisotropies; i++)
			{

				gradient[icell] -= 2.0 * regionval[regions[icell]].anisotropy_magnitudes[i] / regionval[regions[icell]].Ms * regionval[regions[icell]].anisotropy_normals[i] * regionval[regions[icell]].anisotropy_normals[i].dot(spins[icell]);
				//gradient[ispin] = -2.0 * anisotropy_magnitude / Ms * anisotropy_normal * (anisotropy_normal[0]*spins[ispin][0]+anisotropy_normal[1]*spins[ispin][1]+anisotropy_normal[2]*spins[ispin][2]);
				//gradient[ispin] -= 2.0 * 500000 / Ms * temp2 * temp2.dot(spins[ispin]);
				//gradient[ispin] += 2.0 * anisotropy_mag / Ms * ((pow(temp2.dot(spins[ispin]),2)+ pow(temp3.dot(spins[ispin]), 2))*(temp1.dot(spins[ispin])*temp1)+ (pow(temp1.dot(spins[ispin]), 2) + pow(temp3.dot(spins[ispin]), 2))*(temp2.dot(spins[ispin])*temp2)+(pow(temp1.dot(spins[ispin]),2)+ pow(temp2.dot(spins[ispin]), 2))*(temp3.dot(spins[ispin])*temp3));
				//gradient[ispin] -= 2.0 * 500000 / Ms * ((pow(temp2.dot(spins[ispin]), 2) + pow(temp3.dot(spins[ispin]), 2))*(temp1.dot(spins[ispin])*temp1) + (pow(temp1.dot(spins[ispin]), 2) + pow(temp3.dot(spins[ispin]), 2))*(temp2.dot(spins[ispin])*temp2));

			}
		}
	}

	void Hamiltonian_Micromagnetic::Gradient_Anisotropy(const vectorfield & spins, vectorfield & gradient)
	{
		int size = geometry->n_cells_total;
		CU_Gradient_Anisotropy1 << <(size + 1023) / 1024, 1024 >> > (spins.data(), this->geometry->atom_types.data(), this->geometry->n_cell_atoms, gradient.data(), size, regions_book.data(), regions.data());
		CU_CHECK_AND_SYNC();
	}

	__global__ void CU_Gradient_Exchange1(const Vector3 * spins, const int * atom_types, const int * boundary_conditions, const int * n_cells, int n_cell_atoms,
		int n_pairs, const Pair * neigh, Vector3 * gradient, size_t size, bool A_is_nondiagonal, Matrix3 * exchange_tensors, const Vector3 cell_lengths, const Regionvalues * regionval, const int * regions)
	{
		int bc[3] = { boundary_conditions[0],boundary_conditions[1],boundary_conditions[2] };
		int nc[3] = { n_cells[0],n_cells[1],n_cells[2] };

		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
			icell < size;
			icell += blockDim.x * gridDim.x)
		{
			int ispin = icell;//basically id of a cell
			#ifndef SPIRIT_LOW_MEMORY
				gradient[icell][0]=0;
				gradient[icell][1]=0;
				gradient[icell][2]=0;
			#endif
			for (unsigned int i = 0; i < 3; ++i)
			{

				int ispin_plus = cu_idx_from_pair(ispin, bc, nc, n_cell_atoms, atom_types, neigh[2 * i]);
				int ispin_minus = cu_idx_from_pair(ispin, bc, nc, n_cell_atoms, atom_types, neigh[2 * i + 1]);
				if (ispin_plus == -1) {
					ispin_plus = ispin;
				}
				if (ispin_minus == -1) {
					ispin_minus = ispin;
				}

				gradient[ispin][0] -= 2 * exchange_tensors[ispin](i, i)/regionval[regions[ispin]].Ms * (spins[ispin_plus][0] - 2 * spins[ispin][0] + spins[ispin_minus][0]) / (cell_lengths[i]) / (cell_lengths[i]);
				gradient[ispin][1] -= 2 * exchange_tensors[ispin](i, i)/regionval[regions[ispin]].Ms * (spins[ispin_plus][1] - 2 * spins[ispin][1] + spins[ispin_minus][1]) / (cell_lengths[i]) / (cell_lengths[i]);
				gradient[ispin][2] -= 2 * exchange_tensors[ispin](i, i)/regionval[regions[ispin]].Ms * (spins[ispin_plus][2] - 2 * spins[ispin][2] + spins[ispin_minus][2]) / (cell_lengths[i]) / (cell_lengths[i]);

			}
			/*if (A_is_nondiagonal == true) {
				//xy
				int ispin_right = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[0]);
				int ispin_left = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[1]);
				int ispin_top = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[2]);
				int ispin_bottom = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[3]);

				if (ispin_right == -1) {
					ispin_right = ispin;
				}
				if (ispin_left == -1) {
					ispin_left = ispin;
				}
				if (ispin_top == -1) {
					ispin_top = ispin;
				}
				if (ispin_bottom == -1) {
					ispin_bottom = ispin;
				}
				gradient[ispin][0] -= 2 * exchange_tensor(0, 1) / Ms * ((spatial_gradient[ispin_top](0, 0) - spatial_gradient[ispin_bottom](0, 0)) / 4 / cell_lengths[1] + (spatial_gradient[ispin_right](0, 1) - spatial_gradient[ispin_left](0, 1)) / 4 / cell_lengths[0]);
				gradient[ispin][0] -= 2 * exchange_tensor(1, 0) / Ms * ((spatial_gradient[ispin_top](0, 0) - spatial_gradient[ispin_bottom](0, 0)) / 4 / cell_lengths[1] + (spatial_gradient[ispin_right](0, 1) - spatial_gradient[ispin_left](0, 1)) / 4 / cell_lengths[0]);
				gradient[ispin][1] -= 2 * exchange_tensor(0, 1) / Ms * ((spatial_gradient[ispin_top](1, 0) - spatial_gradient[ispin_bottom](1, 0)) / 4 / cell_lengths[1] + (spatial_gradient[ispin_right](1, 1) - spatial_gradient[ispin_left](1, 1)) / 4 / cell_lengths[0]);
				gradient[ispin][1] -= 2 * exchange_tensor(1, 0) / Ms * ((spatial_gradient[ispin_top](1, 0) - spatial_gradient[ispin_bottom](1, 0)) / 4 / cell_lengths[1] + (spatial_gradient[ispin_right](1, 1) - spatial_gradient[ispin_left](1, 1)) / 4 / cell_lengths[0]);
				gradient[ispin][2] -= 2 * exchange_tensor(0, 1) / Ms * ((spatial_gradient[ispin_top](2, 0) - spatial_gradient[ispin_bottom](2, 0)) / 4 / cell_lengths[1] + (spatial_gradient[ispin_right](2, 1) - spatial_gradient[ispin_left](2, 1)) / 4 / cell_lengths[0]);
				gradient[ispin][2] -= 2 * exchange_tensor(1, 0) / Ms * ((spatial_gradient[ispin_top](2, 0) - spatial_gradient[ispin_bottom](2, 0)) / 4 / cell_lengths[1] + (spatial_gradient[ispin_right](2, 1) - spatial_gradient[ispin_left](2, 1)) / 4 / cell_lengths[0]);

				//xz
				ispin_right = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[0]);
				ispin_left = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[1]);
				ispin_top = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[4]);
				ispin_bottom = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[5]);

				if (ispin_right == -1) {
					ispin_right = ispin;
				}
				if (ispin_left == -1) {
					ispin_left = ispin;
				}
				if (ispin_top == -1) {
					ispin_top = ispin;
				}
				if (ispin_bottom == -1) {
					ispin_bottom = ispin;
				}
				gradient[ispin][0] -= 2 * exchange_tensor(0, 2) / Ms * ((spatial_gradient[ispin_top](0, 0) - spatial_gradient[ispin_bottom](0, 0)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](0, 2) - spatial_gradient[ispin_left](0, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][0] -= 2 * exchange_tensor(2, 0) / Ms * ((spatial_gradient[ispin_top](0, 0) - spatial_gradient[ispin_bottom](0, 0)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](0, 2) - spatial_gradient[ispin_left](0, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][1] -= 2 * exchange_tensor(0, 2) / Ms * ((spatial_gradient[ispin_top](1, 0) - spatial_gradient[ispin_bottom](1, 0)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](1, 2) - spatial_gradient[ispin_left](1, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][1] -= 2 * exchange_tensor(2, 0) / Ms * ((spatial_gradient[ispin_top](1, 0) - spatial_gradient[ispin_bottom](1, 0)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](1, 2) - spatial_gradient[ispin_left](1, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][2] -= 2 * exchange_tensor(0, 2) / Ms * ((spatial_gradient[ispin_top](2, 0) - spatial_gradient[ispin_bottom](2, 0)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](2, 2) - spatial_gradient[ispin_left](2, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][2] -= 2 * exchange_tensor(2, 0) / Ms * ((spatial_gradient[ispin_top](2, 0) - spatial_gradient[ispin_bottom](2, 0)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](2, 2) - spatial_gradient[ispin_left](2, 2)) / 4 / cell_lengths[0]);

				//yz
				ispin_right = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[2]);
				ispin_left = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[3]);
				ispin_top = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[4]);
				ispin_bottom = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[5]);

				if (ispin_right == -1) {
					ispin_right = ispin;
				}
				if (ispin_left == -1) {
					ispin_left = ispin;
				}
				if (ispin_top == -1) {
					ispin_top = ispin;
				}
				if (ispin_bottom == -1) {
					ispin_bottom = ispin;
				}
				gradient[ispin][0] -= 2 * exchange_tensor(1, 2) / Ms * ((spatial_gradient[ispin_top](0, 1) - spatial_gradient[ispin_bottom](0, 1)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](0, 2) - spatial_gradient[ispin_left](0, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][0] -= 2 * exchange_tensor(2, 1) / Ms * ((spatial_gradient[ispin_top](0, 1) - spatial_gradient[ispin_bottom](0, 1)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](0, 2) - spatial_gradient[ispin_left](0, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][1] -= 2 * exchange_tensor(1, 2) / Ms * ((spatial_gradient[ispin_top](1, 1) - spatial_gradient[ispin_bottom](1, 1)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](1, 2) - spatial_gradient[ispin_left](1, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][1] -= 2 * exchange_tensor(2, 1) / Ms * ((spatial_gradient[ispin_top](1, 1) - spatial_gradient[ispin_bottom](1, 1)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](1, 2) - spatial_gradient[ispin_left](1, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][2] -= 2 * exchange_tensor(1, 2) / Ms * ((spatial_gradient[ispin_top](2, 1) - spatial_gradient[ispin_bottom](2, 1)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](2, 2) - spatial_gradient[ispin_left](2, 2)) / 4 / cell_lengths[0]);
				gradient[ispin][2] -= 2 * exchange_tensor(2, 1) / Ms * ((spatial_gradient[ispin_top](2, 1) - spatial_gradient[ispin_bottom](2, 1)) / 4 / cell_lengths[2] + (spatial_gradient[ispin_right](2, 2) - spatial_gradient[ispin_left](2, 2)) / 4 / cell_lengths[0]);

			}*/

		}
	}
	void Hamiltonian_Micromagnetic::Gradient_Exchange(const vectorfield & spins, vectorfield & gradient)
	{
		int size = geometry->n_cells_total;
		CU_Gradient_Exchange1 << <(size + 1023) / 1024, 1024 >> > (spins.data(), this->geometry->atom_types.data(), boundary_conditions.data(), geometry->n_cells.data(), geometry->n_cell_atoms,
			this->neigh.size(), this->neigh.data(), gradient.data(), size, A_is_nondiagonal, exchange_tensors.data(),cell_sizes, regions_book.data(), regions.data());
		CU_CHECK_AND_SYNC();
	}

	__global__ void CU_Spatial_Gradient(const Vector3 * spins, const int * atom_types, const int * boundary_conditions, const int * n_cells, int n_cell_atoms,
		int n_pairs, const Pair * neigh, Matrix3 * spatial_gradient, size_t size, const Vector3 cell_lengths)
	{

		/*
		dn1/dr1 dn1/dr2 dn1/dr3
		dn2/dr1 dn2/dr2 dn2/dr3
		dn3/dr1 dn3/dr2 dn3/dr3
		*/
		int bc[3] = { boundary_conditions[0],boundary_conditions[1],boundary_conditions[2] };
		int nc[3] = { n_cells[0],n_cells[1],n_cells[2] };
		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
			icell < size;
			icell += blockDim.x * gridDim.x)
		{
			int ispin = icell;//basically id of a cell
			for (unsigned int i = 0; i < 3; ++i)
			{
				int ispin_plus = cu_idx_from_pair(ispin, bc,nc, n_cell_atoms, atom_types, neigh[2 * i]);
				int ispin_minus = cu_idx_from_pair(ispin, bc, nc, n_cell_atoms, atom_types, neigh[2 * i + 1]);
				if (ispin_plus == -1) {
					ispin_plus = ispin;
				}
				if (ispin_minus == -1) {
					ispin_minus = ispin;
				}
				spatial_gradient[ispin](0, i) = (spins[ispin_plus][0] - spins[ispin_minus][0]) / (cell_lengths[i]) / 2;
				spatial_gradient[ispin](1, i) = (spins[ispin_plus][1] - spins[ispin_minus][1]) / (cell_lengths[i]) / 2;
				spatial_gradient[ispin](2, i) = (spins[ispin_plus][2] - spins[ispin_minus][2]) / (cell_lengths[i]) / 2;

			}
		}
	}
	void Hamiltonian_Micromagnetic::Spatial_Gradient(const vectorfield & spins)
	{
		int size = geometry->n_cells_total;
		CU_Spatial_Gradient << <(size + 1023) / 1024, 1024 >> > (spins.data(), this->geometry->atom_types.data(), boundary_conditions.data(), geometry->n_cells.data(), geometry->n_cell_atoms,
			this->neigh.size(), this->neigh.data(), spatial_gradient.data(), size, cell_sizes);
		CU_CHECK_AND_SYNC();
	}

	__global__ void CU_Gradient_DMI1(const Vector3 * spins, Vector3 * gradient, const Matrix3 * spatial_gradient, size_t size, const Regionvalues * regionval, const int * regions)
	{
		Matrix3 dmi_tensor;
		dmi_tensor(0,0)=0;
		dmi_tensor(0,1)=0;
		dmi_tensor(0,2)=0;
		dmi_tensor(1,0)=0;
		dmi_tensor(1,1)=0;
		dmi_tensor(1,2)=0;
		dmi_tensor(2,0)=0;
		dmi_tensor(2,1)=0;
		dmi_tensor(2,2)=0;
		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
			icell < size;
			icell += blockDim.x * gridDim.x)
		{
			int ispin = icell;//basically id of a cell
			dmi_tensor(0,1)=-regionval[regions[ispin]].Dmi;
			dmi_tensor(1,0)=regionval[regions[ispin]].Dmi;
			#ifndef SPIRIT_LOW_MEMORY
				gradient[icell][0]=0;
				gradient[icell][1]=0;
				gradient[icell][2]=0;
			#endif
			for (unsigned int i = 0; i < 3; ++i)
			{
				gradient[ispin][0] += 2 * dmi_tensor(1, i) /regionval[regions[ispin]].Ms * spatial_gradient[ispin](2, i) - 2 * dmi_tensor(2, i) /regionval[regions[ispin]].Ms * spatial_gradient[ispin](1, i);
				gradient[ispin][1] += 2 * dmi_tensor(2, i) /regionval[regions[ispin]].Ms * spatial_gradient[ispin](0, i) - 2 * dmi_tensor(0, i) /regionval[regions[ispin]].Ms * spatial_gradient[ispin](2, i);
				gradient[ispin][2] += 2 * dmi_tensor(0, i) /regionval[regions[ispin]].Ms * spatial_gradient[ispin](1, i) - 2 * dmi_tensor(1, i) /regionval[regions[ispin]].Ms * spatial_gradient[ispin](0, i);
			}
		}
	}
	void Hamiltonian_Micromagnetic::Gradient_DMI(const vectorfield & spins, vectorfield & gradient)
	{
		int size = geometry->n_cells_total;
		CU_Gradient_DMI1 << <(size + 1023) / 1024, 1024 >> > (spins.data(), gradient.data(), spatial_gradient.data(), size, regions_book.data(), regions.data());
		CU_CHECK_AND_SYNC();
	}
	double g(double x, double y, double z) {
		double R = sqrt(x*x + y * y + z * z);
		return (x*y*z*asinh(z / sqrt(x*x + y * y)) + (y / 6)*(3 * z*z - y * y)*asinh(x / sqrt(y*y + z * z)) + (x / 6)*(3 * z*z - x * x)*asinh(y / sqrt(x*x + z * z)) - (z*z*z / 6)*atan(x*y / z / R) - (z*y*y / 2)*atan(x*z / y / R) - (z*x*x / 2)*atan(y*z / x / R) - x * y*R / 3);
	}
	double f(double x, double y, double z) {
		double R = sqrt(x*x + y * y + z * z);
		return (y / 2)*(z*z - x * x)*asinh(y / sqrt(x*x + z * z)) + (z / 2)*(y*y - x * x)*asinh(z / sqrt(x*x + y * y)) - x * y*z*atan(y*z / x / R) + 1 / 6 * (2 * x*x - y * y - z * z)*R;
	}
	double F2(double x, double y, double z) {
		return f(x, y, z) - f(x, 0, z) - f(x, y, 0) + f(x, 0, 0);
	}
	double F1(double x, double y, double z, double dx, double dy, double dz) {
		return F2(x, y, z) - F2(x, y - dy, z) - F2(x, y, z - dz) + F2(x, y - dy, z - dz);
	}
	double F0(double x, double y, double z, double dx, double dy, double dz) {
		return F1(x, y + dy, z + dz, dx, dy, dz) - F1(x, y, z + dz, dx, dy, dz) - F1(x, y + dy, z, dx, dy, dz) + F1(x, y, z, dx, dy, dz);
	}
	double Nii(double x, double y, double z, double dx, double dy, double dz) {
		double pi = 3.1415926535f;
		return 1 / (4 * pi*dx*dy*dz)*(2 * F0(x, y, z, dx, dy, dz) - F0(x + dx, y, z, dx, dy, dz) - F0(x - dx, y, z, dx, dy, dz));
	}
	double G2(double x, double y, double z) {
		return g(x, y, z) - g(x, y, 0);
	}
	double G1(double x, double y, double z, double dx, double dy, double dz) {
		return G2(x + dx, y, z + dz) - G2(x + dz, y, z) - G2(x, y, z + dz) + G2(x, y, z);
	}
	double G0(double x, double y, double z, double dx, double dy, double dz) {
		return G1(x, y, z, dx, dy, dz) - G1(x, y - dy, z, dx, dy, dz) - G1(x, y, z - dz, dx, dy, dz) + G1(x, y - dy, z - dz, dx, dy, dz);
	}
	double Nij(double x, double y, double z, double dx, double dy, double dz) {
		double pi = 3.1415926535f;
		return 1 / (4 * pi*dx*dy*dz)*(G0(x, y, z, dx, dy, dz) - G0(x - dx, y, z, dx, dy, dz) - G0(x, y + dy, z, dx, dy, dz) + G0(x - dx, y + dy, z, dx, dy, dz));
	}
	double r(int I,int J, int K,int i, int k, int j, double dx, double dy, double dz){	 
		return sqrt((I + i - 0.5f)*(I + i - 0.5f)*dx*dx + (J + j - 0.5f)*(J + j - 0.5f)*dy*dy + (K + k - 0.5f)*(K + k - 0.5f)*dz*dz);
	}
	


	__global__ void CU_FFT_Pointwise_Mult1(FFT::FFT_cpx_type * ft_D_matrices, FFT::FFT_cpx_type * ft_spins, FFT::FFT_cpx_type * res_mult, int* iteration_bounds, int i_b1, int* inter_sublattice_lookup, FFT::StrideContainer dipole_stride, FFT::StrideContainer spin_stride)
	{
		int n = iteration_bounds[0] * iteration_bounds[1] * iteration_bounds[2] * iteration_bounds[3];
		int tupel[4];
		int idx_b1, idx_b2, idx_d;

		for (int ispin = blockIdx.x * blockDim.x + threadIdx.x; ispin < n; ispin += blockDim.x * gridDim.x)
		{
			cu_tupel_from_idx(ispin, tupel, iteration_bounds, 4); // tupel now is {i_b2, a, b, c}

			int& b_inter = inter_sublattice_lookup[i_b1 + tupel[0] * iteration_bounds[0]];

			idx_b1 = i_b1 * spin_stride.basis + tupel[1] * spin_stride.a + tupel[2] * spin_stride.b + tupel[3] * spin_stride.c;
			idx_b2 = tupel[0] * spin_stride.basis + tupel[1] * spin_stride.a + tupel[2] * spin_stride.b + tupel[3] * spin_stride.c;
			idx_d = b_inter * dipole_stride.basis + tupel[1] * dipole_stride.a + tupel[2] * dipole_stride.b + tupel[3] * dipole_stride.c;

			auto& fs_x = ft_spins[idx_b2];
			auto& fs_y = ft_spins[idx_b2 + 1 * spin_stride.comp];
			auto& fs_z = ft_spins[idx_b2 + 2 * spin_stride.comp];

			auto& fD_xx = ft_D_matrices[idx_d];
			auto& fD_xy = ft_D_matrices[idx_d + 1 * dipole_stride.comp];
			auto& fD_xz = ft_D_matrices[idx_d + 2 * dipole_stride.comp];
			auto& fD_yy = ft_D_matrices[idx_d + 3 * dipole_stride.comp];
			auto& fD_yz = ft_D_matrices[idx_d + 4 * dipole_stride.comp];
			auto& fD_zz = ft_D_matrices[idx_d + 5 * dipole_stride.comp];

			if (tupel[0] == 0)
			{
				res_mult[idx_b1].x = FFT::mult3D(fD_xx, fD_xy, fD_xz, fs_x, fs_y, fs_z).x;
				res_mult[idx_b1].y = FFT::mult3D(fD_xx, fD_xy, fD_xz, fs_x, fs_y, fs_z).y;
				res_mult[idx_b1 + 1 * spin_stride.comp].x = FFT::mult3D(fD_xy, fD_yy, fD_yz, fs_x, fs_y, fs_z).x;
				res_mult[idx_b1 + 1 * spin_stride.comp].y = FFT::mult3D(fD_xy, fD_yy, fD_yz, fs_x, fs_y, fs_z).y;
				res_mult[idx_b1 + 2 * spin_stride.comp].x = FFT::mult3D(fD_xz, fD_yz, fD_zz, fs_x, fs_y, fs_z).x;
				res_mult[idx_b1 + 2 * spin_stride.comp].y = FFT::mult3D(fD_xz, fD_yz, fD_zz, fs_x, fs_y, fs_z).y;
			}
			else {
				atomicAdd(&res_mult[idx_b1].x, FFT::mult3D(fD_xx, fD_xy, fD_xz, fs_x, fs_y, fs_z).x);
				atomicAdd(&res_mult[idx_b1].y, FFT::mult3D(fD_xx, fD_xy, fD_xz, fs_x, fs_y, fs_z).y);
				atomicAdd(&res_mult[idx_b1 + 1 * spin_stride.comp].x, FFT::mult3D(fD_xy, fD_yy, fD_yz, fs_x, fs_y, fs_z).x);
				atomicAdd(&res_mult[idx_b1 + 1 * spin_stride.comp].y, FFT::mult3D(fD_xy, fD_yy, fD_yz, fs_x, fs_y, fs_z).y);
				atomicAdd(&res_mult[idx_b1 + 2 * spin_stride.comp].x, FFT::mult3D(fD_xz, fD_yz, fD_zz, fs_x, fs_y, fs_z).x);
				atomicAdd(&res_mult[idx_b1 + 2 * spin_stride.comp].y, FFT::mult3D(fD_xz, fD_yz, fD_zz, fs_x, fs_y, fs_z).y);
			}
		}
	}

	__global__ void CU_Write_FFT_Gradients1(const FFT::FFT_real_type * resiFFT, Vector3 * gradient, FFT::StrideContainer spin_stride, int * iteration_bounds, int n_cell_atoms, scalar * mu_s, int sublattice_size, const scalar minMs, const Regionvalues * regionval, const int * regions)
	{
		int nos = iteration_bounds[0] * iteration_bounds[1] * iteration_bounds[2] * iteration_bounds[3];
		int tupel[4];
		int idx_pad;
		for (int idx_orig = blockIdx.x * blockDim.x + threadIdx.x; idx_orig < nos; idx_orig += blockDim.x * gridDim.x)
		{

			cu_tupel_from_idx(idx_orig, tupel, iteration_bounds, 4); //tupel now is {ib, a, b, c}
			idx_pad = tupel[0] * spin_stride.basis + tupel[1] * spin_stride.a + tupel[2] * spin_stride.b + tupel[3] * spin_stride.c;
			//printf("%d %f %f\n", idx_orig, resiFFT[this->idx_pad],gradient[this->idx_orig][0]);
			#ifndef SPIRIT_LOW_MEMORY
				gradient[idx_orig][0] = -resiFFT[idx_pad]*minMs*1e-7/(sublattice_size);
				gradient[idx_orig][1] = -resiFFT[idx_pad + 1 * spin_stride.comp]*minMs*1e-7/(sublattice_size);
				gradient[idx_orig][2] = -resiFFT[idx_pad + 2 * spin_stride.comp]*minMs*1e-7/(sublattice_size);
			#endif
			#ifdef SPIRIT_LOW_MEMORY
				gradient[idx_orig][0] -= resiFFT[idx_pad]*minMs*1e-7/(sublattice_size);
				gradient[idx_orig][1] -= resiFFT[idx_pad + 1 * spin_stride.comp]*minMs*1e-7/(sublattice_size);
				gradient[idx_orig][2] -= resiFFT[idx_pad + 2 * spin_stride.comp]*minMs*1e-7/(sublattice_size);
			#endif
		}
	}

	void Hamiltonian_Micromagnetic::Gradient_DDI(const vectorfield & spins, vectorfield & gradient)
	{
		//this->Gradient_DDI_Direct(spins, gradient);
		this->Gradient_DDI_FFT(spins, gradient);
		/*
		if (this->ddi_method == DDI_Method::FFT)
		{
			printf("sasas");
			this->Gradient_DDI_FFT(spins, gradient);
		}
			else if (this->ddi_method == DDI_Method::Cutoff)
			{
				// TODO: Merge these implementations in the future
				if (this->ddi_cutoff_radius >= 0)
					this->Gradient_DDI_Cutoff(spins, gradient);
				else
					this->Gradient_DDI_Direct(spins, gradient);
			}
*/
	}
	void Hamiltonian_Micromagnetic::Gradient_DDI_Cutoff(const vectorfield & spins, vectorfield & gradient)
		{
			// TODO
		}
	__global__ void CU_Mult_Spins(const scalar minMs, const int n_cells_total, const Regionvalues * regionval, const int * regions, const Vector3 * spins, Vector3 * mult_spins)
	{
		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
					icell < n_cells_total;
					icell += blockDim.x * gridDim.x)
		{
			mult_spins[icell]=spins[icell]*regionval[regions[icell]].Ms/minMs;
		}
	}
	__global__ void CU_Div_Spins(const scalar minMs, const int n_cells_total, const Regionvalues * regionval, const int * regions, const Vector3 * spins, Vector3 * mult_spins)
	{
		for (auto icell = blockIdx.x * blockDim.x + threadIdx.x;
					icell < n_cells_total;
					icell += blockDim.x * gridDim.x)
		{
			mult_spins[icell]=spins[icell]/regionval[regions[icell]].Ms*minMs;
		}
	}
	void Hamiltonian_Micromagnetic::Gradient_DDI_FFT(const vectorfield & spins, vectorfield & gradient)
		{
			auto& ft_D_matrices = transformed_dipole_matrices;

			auto& ft_spins = fft_plan_spins.cpx_ptr;

			auto& res_iFFT = fft_plan_reverse.real_ptr;
			auto& res_mult = fft_plan_reverse.cpx_ptr;

			int number_of_mults = it_bounds_pointwise_mult[0] * it_bounds_pointwise_mult[1] * it_bounds_pointwise_mult[2] * it_bounds_pointwise_mult[3];
			//#ifndef SPIRIT_LOW_MEMORY
			CU_Mult_Spins << <(geometry->nos + 1023) / 1024, 1024 >> > (minMs, geometry->n_cells_total, regions_book.data(),regions.data(),spins.data(),mult_spins.data());
			CU_CHECK_AND_SYNC();
			FFT_Spins(mult_spins);
			/*#endif
			#ifdef SPIRIT_LOW_MEMORY
				CU_Mult_Spins << <(geometry->nos + 1023) / 1024, 1024 >> > (minMs, geometry->n_cells_total, regions_book.data(),regions.data(),spins.data(), spins.data());
				CU_CHECK_AND_SYNC();
				FFT_Spins(spins);
			#endif*/
			//FFT_Spins(spins);

			// TODO: also parallelize over i_b1
			// Loop over basis atoms (i.e sublattices) and add contribution of each sublattice
			for (int i_b1 = 0; i_b1 < geometry->n_cell_atoms; ++i_b1)
				CU_FFT_Pointwise_Mult1 << <(number_of_mults + 1023) / 1024, 1024 >> > (ft_D_matrices.data(), ft_spins.data(), res_mult.data(), it_bounds_pointwise_mult.data(), i_b1, inter_sublattice_lookup.data(), dipole_stride, spin_stride);
				CU_CHECK_AND_SYNC();
			FFT::batch_iFour_3D(fft_plan_reverse);

			CU_Write_FFT_Gradients1 << <(geometry->nos + 1023) / 1024, 1024 >> > (res_iFFT.data(), gradient.data(), spin_stride, it_bounds_write_gradients.data(), geometry->n_cell_atoms, geometry->mu_s.data(), sublattice_size, minMs, regions_book.data(), regions.data());
			CU_CHECK_AND_SYNC();

			/*#ifdef SPIRIT_LOW_MEMORY
				CU_Div_Spins << <(geometry->nos + 1023) / 1024, 1024 >> > (minMs, geometry->n_cells_total, regions_book.data(),regions.data(),spins.data(),spins.data());
				CU_CHECK_AND_SYNC();
			#endif*/
		}//end Field_DipoleDipole

	void Hamiltonian_Micromagnetic::Gradient_DDI_Direct(const vectorfield & spins, vectorfield & gradient)
		{
			int tupel1[3];
			int tupel2[3];
			int sublattice_size = it_bounds_write_dipole[0] * it_bounds_write_dipole[1] * it_bounds_write_dipole[2];
					//prefactor of ddi interaction
					//scalar mult = 2.0133545*1e-28 * 0.057883817555 * 0.057883817555 / (4 * 3.141592653589793238462643383279502884197169399375105820974 * 1e-30);
			scalar mult = 1 / (4 * 3.141592653589793238462643383279502884197169399375105820974);
			scalar m0 = (4 * 3.141592653589793238462643383279502884197169399375105820974)*1e-7;
			int img_a = boundary_conditions[0] == 0 ? 0 : ddi_n_periodic_images[0];
			int img_b = boundary_conditions[1] == 0 ? 0 : ddi_n_periodic_images[1];
			int img_c = boundary_conditions[2] == 0 ? 0 : ddi_n_periodic_images[2];

			for (int idx1 = 0; idx1 < geometry->nos; idx1++)
			{
				gradient[idx1][0]=0;
				gradient[idx1][1]=0;
				gradient[idx1][2]=0;
				double kk=0;
				for (int idx2 = 0; idx2 < geometry->nos; idx2++)
				{
					int a1 = idx1%(it_bounds_write_spins[1]);
					int b1 = ((int)(idx1/it_bounds_write_spins[1]))%(it_bounds_write_spins[2]);
					int c1 = (int)idx1/(it_bounds_write_spins[1]*it_bounds_write_spins[2]);
					int a2 = idx2%(it_bounds_write_spins[1]);
					int b2 = ((int)(idx2/it_bounds_write_spins[1]))%(it_bounds_write_spins[2]);
					int c2 = (int)idx2/(it_bounds_write_spins[1]*it_bounds_write_spins[2]);
					/*int a_idx = a < n_cells[0] ? a : a - iteration_bounds[0];
					int b_idx = b < n_cells[1] ? b : b - iteration_bounds[1];
					int c_idx = c < n_cells[2] ? c : c - iteration_bounds[2];*/
					int a_idx = a1-a2;
					int b_idx = b1-b2;
					int c_idx = c1-c2;
					if ((a_idx==0) && (b_idx==0) && (c_idx==0)){
						//printf("test\n");
						//continue;
					}
					//printf("%d %d %d\n", a_idx,b_idx,c_idx);
					/*if ((a_idx==20) || (b_idx==20) || (c_idx==1)){
						//printf("test1\n");
						//if (c_idx!=1)
							//printf("%d %d %d %d\n", a_idx, b_idx, c_idx,  dipole_stride.comp);
						continue;
					}*/
					//scalar delta[3] = { 3,3,0.3 };
					//int idx = b_inter * dipole_stride.basis + a * dipole_stride.a + b * dipole_stride.b + c * dipole_stride.c;
					scalar Dxx = 0, Dxy = 0, Dxz = 0, Dyy = 0, Dyz = 0, Dzz = 0;
					//asa
					for (int i = 0; i < 2; i++) {
						for (int j = 0; j < 2; j++) {
							for (int k = 0; k < 2; k++) {
								double r = sqrt((a_idx + i - 0.5f)*(a_idx + i - 0.5f)*cell_sizes[0]* cell_sizes[0] + (b_idx + j - 0.5f)*(b_idx + j-0.5f)*cell_sizes[1] * cell_sizes[1] + (c_idx + k - 0.5f)*(c_idx + k - 0.5f)*cell_sizes[2] * cell_sizes[2]);
								Dxx += mult * pow(-1.0f, i + j + k) * atan(((c_idx + k-0.5f) * (b_idx + j - 0.5f) * cell_sizes[1]*cell_sizes[2]/cell_sizes[0] / r / (a_idx + i - 0.5f)));
								//fft_dipole_inputs[this->idx + 1 * dipole_stride.comp] += -mult * pow(-1.0f, i + j + k) * log(abs(((c_idx + k - 0.5f)* cell_sizes[2] + r)/((c_idx + k - 0.5f)* cell_sizes[2] - r)));
								//fft_dipole_inputs[this->idx + 2 * dipole_stride.comp] += -mult * pow(-1.0f, i + j + k) * log(abs(((b_idx + j - 0.5f)* cell_sizes[1] + r)/((b_idx + j - 0.5f)* cell_sizes[1] - r)));
								Dxy -= mult * pow(-1.0f, i + j + k) * log((((c_idx + k - 0.5f)* cell_sizes[2] + r)));
								Dxz -= mult * pow(-1.0f, i + j + k) * log((((b_idx + j - 0.5f)* cell_sizes[1] + r)));

								Dyy += mult * pow(-1.0f, i + j + k) * atan(((a_idx + i-0.5f) * (c_idx + k - 0.5f) * cell_sizes[2]*cell_sizes[0]/cell_sizes[1] / r / (b_idx + j - 0.5f)));
								//fft_dipole_inputs[this->idx + 4 * dipole_stride.comp] += -mult * pow(-1.0f, i + j + k) * log(abs(((a_idx + i - 0.5f)* cell_sizes[0] + r)/((a_idx + i - 0.5f)* cell_sizes[0] - r)));
								Dyz -= mult * pow(-1.0f, i + j + k) * log((((a_idx + i - 0.5f)* cell_sizes[0] + r)));
								Dzz += mult * pow(-1.0f, i + j + k) * atan(((b_idx + j-0.5f) * (a_idx + i - 0.5f) * cell_sizes[0]*cell_sizes[1]/cell_sizes[2] / r / (c_idx + k - 0.5f)));

							}
						}
					}/*
					Dxx=Nii(a_idx*delta[0],b_idx*delta[1],c_idx*delta[2],delta[0],delta[1],delta[2]);
					Dxy=Nij(a_idx*delta[0],b_idx*delta[1],c_idx*delta[2],delta[0],delta[1],delta[2]);
					Dxz=Nij(a_idx*delta[0],c_idx*delta[2], b_idx*delta[1],delta[0],delta[2],delta[1]);
					Dyy=Nii(b_idx*delta[1],a_idx*delta[0],c_idx*delta[2],delta[1],delta[0],delta[2]);
					Dyz=Nij(b_idx*delta[1],c_idx*delta[2], b_idx*delta[1],delta[1],delta[2],delta[0]);
					Dzz=Nii(c_idx*delta[2],a_idx*delta[0],b_idx*delta[1],delta[2],delta[0],delta[1]);*/
					if (idx1==42){
						if ((a_idx==0) && (b_idx==0) && (c_idx==0)){
							printf("000 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
						}
						if ((a_idx==1) && (b_idx==0) && (c_idx==0)){
							printf("100 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
						}
						if ((a_idx==0) && (b_idx==1) && (c_idx==0)){
							printf("010 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
						}
						if ((a_idx==-1) && (b_idx==1) && (c_idx==0)){
							printf("-110 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
						}
						if ((a_idx==1) && (b_idx==1) && (c_idx==0)){
							printf("110 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
						}
						if ((a_idx==2) && (b_idx==0) && (c_idx==0)){
							printf("200 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
						}
						if ((a_idx==0) && (b_idx==2) && (c_idx==0)){
							printf("020 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
						}
						if ((a_idx==2) && (b_idx==2) && (c_idx==0)){
													printf("220 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
												}
						if ((a_idx==2) && (b_idx==-2) && (c_idx==0)){
													printf("2-20 Dxx=%f Dxy=%f Dxz=%f Dyy=%f Dyz=%f Dzz=%f\n",Dxx,Dxy,Dxz,Dyy,Dyz, Dzz);
												}
						//printf("x=%f y=%f z=%f\n",spins[this->idx2][0],spins[this->idx2][1],spins[this->idx2][2]);
					}
					kk+=Dxx;
					/*gradient[idx1][0] -= (Dxx * spins[idx2][0] + Dxy * spins[idx2][1] + Dxz * spins[idx2][2]) * Ms*m0;
					gradient[idx1][1] -= (Dxy * spins[idx2][0] + Dyy * spins[idx2][1] + Dyz * spins[idx2][2]) * Ms*m0;
					gradient[idx1][2] -= (Dxz * spins[idx2][0] + Dyz * spins[idx2][1] + Dzz * spins[idx2][2]) * Ms*m0;*/
				}
				if (idx1==30){
					//printf("x=%f y=%f z=%f\n",spins[this->idx1][0],spins[this->idx1][1],spins[this->idx1][2]);
					//printf("kk=%f gx=%f gy=%f gz=%f\n",kk, gradient[this->idx1][0]/8e5/m0,gradient[this->idx1][1],gradient[this->idx1][2]);

				}

			}
		}
	__global__ void CU_Write_FFT_Spin_Input1(FFT::FFT_real_type* fft_spin_inputs, const Vector3 * spins, int * iteration_bounds, FFT::StrideContainer spin_stride, scalar * mu_s)
	{
		int nos = iteration_bounds[0] * iteration_bounds[1] * iteration_bounds[2] * iteration_bounds[3];
		int tupel[4];
		int idx_pad;
		for (int idx_orig = blockIdx.x * blockDim.x + threadIdx.x; idx_orig < nos; idx_orig += blockDim.x * gridDim.x)
		{
			cu_tupel_from_idx(idx_orig, tupel, iteration_bounds, 4); //tupel now is {ib, a, b, c}
			idx_pad = tupel[0] * spin_stride.basis + tupel[1] * spin_stride.a + tupel[2] * spin_stride.b + tupel[3] * spin_stride.c;
			fft_spin_inputs[idx_pad] = spins[idx_orig][0];
			fft_spin_inputs[idx_pad + 1 * spin_stride.comp] = spins[idx_orig][1];
			fft_spin_inputs[idx_pad + 2 * spin_stride.comp] = spins[idx_orig][2];
			//printf("%f %f\n",fft_spin_inputs[this->idx_pad], fft_spin_inputs[this->idx_pad+30]);
		}
	}

	void Hamiltonian_Micromagnetic::FFT_Spins(const vectorfield & spins)
	{
		CU_Write_FFT_Spin_Input1 << <(geometry->nos + 1023) / 1024, 1024 >> > (fft_plan_spins.real_ptr.data(), spins.data(), it_bounds_write_spins.data(), spin_stride, geometry->mu_s.data());
		CU_CHECK_AND_SYNC();
		FFT::batch_Four_3D(fft_plan_spins);
	}
	__global__ void CU_Write_FFT_Dipole_Input1(FFT::FFT_real_type* fft_dipole_inputs, int* iteration_bounds, const Vector3* translation_vectors, int n_cell_atoms, Vector3* cell_atom_translations, int* n_cells, int* inter_sublattice_lookup, int* img, FFT::StrideContainer dipole_stride, const Vector3 cell_lengths)
	{
		int tupel[3];
		int sublattice_size = iteration_bounds[0] * iteration_bounds[1] * iteration_bounds[2];
		//prefactor of ddi interaction
		//scalar mult = 2.0133545*1e-28 * 0.057883817555 * 0.057883817555 / (4 * 3.141592653589793238462643383279502884197169399375105820974 * 1e-30);
		//scalar mult = 1 / (4 * 3.141592653589793238462643383279502884197169399375105820974);
		scalar mult = 1;
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < sublattice_size; i += blockDim.x * gridDim.x)
		{
			cu_tupel_from_idx(i, tupel, iteration_bounds, 3); // tupel now is {a, b, c}
			auto& a = tupel[0];
			auto& b = tupel[1];
			auto& c = tupel[2];
			/*if ((a>198)||(b>198)||(c>198)){
				printf("%d %d %d\n", a,b,c);
			}*/
			/*int a_idx = a < n_cells[0] ? a : a - iteration_bounds[0];
			int b_idx = b < n_cells[1] ? b : b - iteration_bounds[1];
			int c_idx = c < n_cells[2] ? c : c - iteration_bounds[2];*/
			/*int a_idx = a +1 - (int)iteration_bounds[0]/2;
			int b_idx = b +1- (int)iteration_bounds[1]/2;
			int c_idx = c +1- (int)iteration_bounds[2]/2;*/
			int a_idx = a < n_cells[0] ? a : a - iteration_bounds[0];
			int b_idx = b < n_cells[1] ? b : b - iteration_bounds[1];
			int c_idx = c < n_cells[2] ? c : c - iteration_bounds[2];

			int idx = a * dipole_stride.a + b * dipole_stride.b + c * dipole_stride.c;

			for (int i = 0; i < 2; i++) {
				for (int j = 0; j < 2; j++) {
					for (int k = 0; k < 2; k++) {
						double r = sqrt((a_idx + i - 0.5f)*(a_idx + i - 0.5f)*cell_lengths[0]* cell_lengths[0] + (b_idx + j - 0.5f)*(b_idx + j-0.5f)*cell_lengths[1] * cell_lengths[1] + (c_idx + k - 0.5f)*(c_idx + k - 0.5f)*cell_lengths[2] * cell_lengths[2]);
						fft_dipole_inputs[idx] += mult * pow(-1.0f, i + j + k) * atan(((c_idx + k-0.5f) * (b_idx + j - 0.5f) * cell_lengths[1]*cell_lengths[2]/cell_lengths[0] / r / (a_idx + i - 0.5f)));
						//fft_dipole_inputs[idx + 1 * dipole_stride.comp] += -mult * pow(-1.0f, i + j + k) * log(abs(((c_idx + k - 0.5f)* cell_lengths[2] + r)/((c_idx + k - 0.5f)* cell_lengths[2] - r)));
						//fft_dipole_inputs[idx + 2 * dipole_stride.comp] += -mult * pow(-1.0f, i + j + k) * log(abs(((b_idx + j - 0.5f)* cell_lengths[1] + r)/((b_idx + j - 0.5f)* cell_lengths[1] - r)));
						fft_dipole_inputs[idx + 1 * dipole_stride.comp] -= mult * pow(-1.0f, i + j + k) * log((((c_idx + k - 0.5f)* cell_lengths[2] + r)));
						fft_dipole_inputs[idx + 2 * dipole_stride.comp] -= mult * pow(-1.0f, i + j + k) * log((((b_idx + j - 0.5f)* cell_lengths[1] + r)));

						fft_dipole_inputs[idx + 3 * dipole_stride.comp] += mult * pow(-1.0f, i + j + k) * atan(((a_idx + i-0.5f) * (c_idx + k - 0.5f) * cell_lengths[2]*cell_lengths[0]/cell_lengths[1] / r / (b_idx + j - 0.5f)));
						//fft_dipole_inputs[idx + 4 * dipole_stride.comp] += -mult * pow(-1.0f, i + j + k) * log(abs(((a_idx + i - 0.5f)* cell_lengths[0] + r)/((a_idx + i - 0.5f)* cell_lengths[0] - r)));
						fft_dipole_inputs[idx + 4 * dipole_stride.comp] -= mult * pow(-1.0f, i + j + k) * log((((a_idx + i - 0.5f)* cell_lengths[0] + r)));
						fft_dipole_inputs[idx + 5 * dipole_stride.comp] += mult * pow(-1.0f, i + j + k) * atan(((b_idx + j-0.5f) * (a_idx + i - 0.5f) * cell_lengths[0]*cell_lengths[1]/cell_lengths[2] / r / (c_idx + k - 0.5f)));

					}
				}
			}

				//if (fft_dipole_inputs[this->idx]<-0.03)
		}
	}

	void Hamiltonian_Micromagnetic::FFT_Dipole_Matrices(FFT::FFT_Plan & fft_plan_dipole, int img_a, int img_b, int img_c)
	{
		auto& fft_dipole_inputs = fft_plan_dipole.real_ptr;

		field<int> img = {
							img_a,
							img_b,
							img_c
		};

		// Work around to make bravais vectors and cell_atoms available to GPU as they are currently saves as std::vectors and not fields ...
		auto translation_vectors = field<Vector3>();
		auto cell_atom_translations = field<Vector3>();

		for (int i = 0; i < 3; i++)
			translation_vectors.push_back(geometry->lattice_constant * geometry->bravais_vectors[i]);

		for (int i = 0; i < geometry->n_cell_atoms; i++)
			cell_atom_translations.push_back(geometry->positions[i]);

		CU_Write_FFT_Dipole_Input1 << <(sublattice_size + 1023) / 1024, 1024 >> >
			(fft_dipole_inputs.data(), it_bounds_write_dipole.data(), translation_vectors.data(),
				geometry->n_cell_atoms, cell_atom_translations.data(), geometry->n_cells.data(),
				inter_sublattice_lookup.data(), img.data(), dipole_stride, cell_sizes
				);
		CU_CHECK_AND_SYNC();
		FFT::batch_Four_3D(fft_plan_dipole);
	}
	void Hamiltonian_Micromagnetic::Prepare_DDI()
	{
		Clean_DDI();
		mult_spins=vectorfield(geometry->nos,Vector3{0,0,1});
		minMs=regions_book[0].Ms;
		for(int i=0;i<region_num;i++){
			if (regions_book[i].Ms<minMs) minMs=regions_book[i].Ms;
		}
		n_cells_padded.resize(3);
		n_cells_padded[0] = (geometry->n_cells[0] > 1) ? 2 * geometry->n_cells[0] : 1;
		n_cells_padded[1] = (geometry->n_cells[1] > 1) ? 2 * geometry->n_cells[1] : 1;
		n_cells_padded[2] = (geometry->n_cells[2] > 1) ? 2 * geometry->n_cells[2] : 1;
		sublattice_size = n_cells_padded[0] * n_cells_padded[1] * n_cells_padded[2];
		//printf("111 %d %d %d\n", n_cells_padded[0],n_cells_padded[1],n_cells_padded[2]);

		inter_sublattice_lookup.resize(geometry->n_cell_atoms * geometry->n_cell_atoms);

		//we dont need to transform over length 1 dims
		std::vector<int> fft_dims;
		for (int i = 2; i >= 0; i--) //notice that reverse order is important!
		{
			if (n_cells_padded[i] > 1)
				fft_dims.push_back(n_cells_padded[i]);
		}

		//Count how many distinct inter-lattice contributions we need to store
		n_inter_sublattice = 0;
		for (int i = 0; i < geometry->n_cell_atoms; i++)
		{
			for (int j = 0; j < geometry->n_cell_atoms; j++)
			{
				if (i != 0 && i == j) continue;
				n_inter_sublattice++;
			}
		}
		//printf("lex%d %d %d\n", n_inter_sublattice, fft_dims[0],fft_dims[1]);
		//Set the iteration bounds for the nested for loops that are flattened in the kernels
		it_bounds_write_spins = { geometry->n_cell_atoms,
									  geometry->n_cells[0],
									  geometry->n_cells[1],
									  geometry->n_cells[2] };

		it_bounds_write_dipole = { n_cells_padded[0],
									  n_cells_padded[1],
									  n_cells_padded[2] };

		it_bounds_pointwise_mult = { geometry->n_cell_atoms,
									  (n_cells_padded[0] / 2 + 1), // due to redundancy in real fft
									  n_cells_padded[1],
									  n_cells_padded[2] };

		it_bounds_write_gradients = { geometry->n_cell_atoms,
									  geometry->n_cells[0],
									  geometry->n_cells[1],
									  geometry->n_cells[2] };

		FFT::FFT_Plan fft_plan_dipole = FFT::FFT_Plan(fft_dims, false, 6 * n_inter_sublattice, sublattice_size);
		fft_plan_spins = FFT::FFT_Plan(fft_dims, false, 3 * geometry->n_cell_atoms, sublattice_size);
		fft_plan_reverse = FFT::FFT_Plan(fft_dims, true, 3 * geometry->n_cell_atoms, sublattice_size);

		field<int*> temp_s = { &spin_stride.comp, &spin_stride.basis, &spin_stride.a, &spin_stride.b, &spin_stride.c };
		field<int*> temp_d = { &dipole_stride.comp, &dipole_stride.basis, &dipole_stride.a, &dipole_stride.b, &dipole_stride.c };;
		FFT::get_strides(temp_s, { 3, this->geometry->n_cell_atoms, n_cells_padded[0], n_cells_padded[1], n_cells_padded[2] });
		FFT::get_strides(temp_d, { 6, n_inter_sublattice, n_cells_padded[0], n_cells_padded[1], n_cells_padded[2] });
		/*
		//perform FFT of dipole matrices
		int img_a = boundary_conditions[0] == 0 ? 0 : ddi_n_periodic_images[0];
		int img_b = boundary_conditions[1] == 0 ? 0 : ddi_n_periodic_images[1];
		int img_c = boundary_conditions[2] == 0 ? 0 : ddi_n_periodic_images[2];
	
		FFT_Dipole_Matrices(fft_plan_dipole, img_a, img_b, img_c); */
		FFT_Dipole_Matrices(fft_plan_dipole, 0, 0, 0);

		transformed_dipole_matrices = std::move(fft_plan_dipole.cpx_ptr);
	}//end prepare
	void Hamiltonian_Micromagnetic::Prepare_Exchange()
	{
		//int bc[3] = { boundary_conditions[0],boundary_conditions[1],boundary_conditions[2] };
		//int nc[3] = { geometry->n_cells[0],geometry->n_cells[1],geometry->n_cells[2] };
		exchange_tensors=matrixfield(geometry->nos, Matrix3::Zero());
		for (int ispin=0; ispin<geometry->nos; ispin++){
			for (unsigned int i = 0; i < 3; ++i)
				{
					int ispin_plus = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[2 * i]);
					int ispin_minus = idx_from_pair(ispin, boundary_conditions, geometry->n_cells, geometry->n_cell_atoms, geometry->atom_types, neigh[2 * i + 1]);
					if (ispin_plus == -1) {
						ispin_plus = ispin;
					}
					if (ispin_minus == -1) {
						ispin_minus = ispin;
					}
					exchange_tensors[ispin](i,i)=(exchange_table[regions[ispin]][regions[ispin_minus]]+exchange_table[regions[ispin]][regions[ispin_plus]])/2;
				}
		}
	}
	void Hamiltonian_Micromagnetic::Clean_DDI()
	{
		fft_plan_spins = FFT::FFT_Plan();
		fft_plan_reverse = FFT::FFT_Plan();
	}
	
    void Hamiltonian_Micromagnetic::Hessian(const vectorfield & spins, MatrixX & hessian)
    {
    }


    // Hamiltonian name as string
    static const std::string name = "Micromagnetic";
    const std::string& Hamiltonian_Micromagnetic::Name() { return name; }
}

#endif
